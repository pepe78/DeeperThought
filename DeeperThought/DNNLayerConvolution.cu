#include "hip/hip_runtime.h"
#include "DNNLayerConvolution.cuh"

#include <cstdlib>
#include <cstdio>

#define MAXX1X2 784
#define MAXNUMCONVY1Y2 196

__global__ void convolution_forward(float *outp, const float *inp, const float *pars, int numPics, int inputWidth, int outputWidth, int numConvolutions, int x1, int x2, int y1, int y2, int batchSize)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	if (tid < batchSize)
	{
		float pics[MAXX1X2];
		for (int i = 0; i < x1 * x2 * numPics; i++)
		{
			pics[i] = inp[tid * inputWidth + i];
		}
		float convos[MAXNUMCONVY1Y2];

		int pos = 0;
		for (int c = 0; c < numConvolutions; c++)
		{
		  for (int i = 0; i < y1 * y2; i++)
		  {
			  convos[i] = pars[c * y1 * y2 + i];
		  }

			for (int p = 0; p < numPics; p++)
			{
				for (int i = 0; i < x1 - y1 + 1; i++)
				{
					for (int j = 0; j < x2 - y2 + 1; j++)
					{
						float tmp = 0;
						for (int k = 0; k < y1; k++)
						{
							for (int l = 0; l < y2; l++)
							{
								tmp += pics[p * x1 * x2 + (i + k) * x2 + (j + l)] * convos[k * y2 + l];
							}
						}
						outp[tid * outputWidth + pos] = tmp;
						pos++;
					}
				}
			}
		}
	}
}

__global__ void convolution_backward(float *dinp, float *dpars, const float *doutp, const float *outp, const float *inp, const float *pars, int numPics, int inputWidth, int outputWidth, int numConvolutions, int x1, int x2, int y1, int y2, int batchSize)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	if (tid < batchSize)
	{
		float pics[MAXX1X2];
		for (int i = 0; i < x1 * x2 * numPics; i++)
		{
			pics[i] = inp[tid * inputWidth + i];
		}
		float convos[MAXNUMCONVY1Y2];

		int pos = 0;
		for (int c = 0; c < numConvolutions; c++)
		{
		  for (int i = 0; i < y1 * y2; i++)
		  {
			  convos[i] = pars[c * y1 * y2 + i];
		  }

			for (int p = 0; p < numPics; p++)
			{
				for (int i = 0; i < x1 - y1 + 1; i++)
				{
					for (int j = 0; j < x2 - y2 + 1; j++)
					{
						float tmp = doutp[tid * outputWidth + pos];
						if (tmp != 0)
						{
							for (int k = 0; k < y1; k++)
							{
								for (int l = 0; l < y2; l++)
								{
									if (dinp != NULL)
									{
										dinp[tid * inputWidth + p * x1 * x2 + (i + k) * x2 + (j + l)] += tmp * convos[k * y2 + l];
									}
									atomicAdd(&(dpars[c * y1 * y2 + k * y2 + l]), tmp * pics[p * x1 * x2 + (i + k) * x2 + (j + l)]);
								}
							}
						}
						pos++;
					}
				}
			}
		}
	}
}

DNNLayerConvolution::DNNLayerConvolution(int _numPics, int _x1, int _x2, int _numConvolutions, int _y1, int _y2, int _batchSize, float _initVal, float _stepSize)
	: DNNLayer(_batchSize, _numPics * _x1 * _x2, _numPics * (_x1 - _y1 + 1) * (_x2 - _y2 + 1) * _numConvolutions, _numConvolutions * _y1 * _y2, _initVal, _stepSize)
{
	x1 = _x1;
	x2 = _x2;
	y1 = _y1;
	y2 = _y2;
	numPics = _numPics;
	numConvolutions = _numConvolutions;

	if (x1 * x2 * numPics > MAXX1X2)
	{
		fprintf(stderr, "Project needs to be recompiled with larger field for convolution layer\n");
		exit(-1);
	}
	if (y1 * y2 > MAXNUMCONVY1Y2)
	{
		fprintf(stderr, "Project needs to be recompiled with larger field for convolution layer\n");
		exit(-1);
	}
}

DNNLayerConvolution::~DNNLayerConvolution()
{

}

void DNNLayerConvolution::Forward(CPUGPUMemory* input)
{
	int threadsPerBlock = 256;
	int numBlocks = ((input->GetSize() / inputWidth) + threadsPerBlock - 1) / threadsPerBlock;
	convolution_forward<<<numBlocks, threadsPerBlock>>>(
		(float*)output->GetGPUMemory(), (float*)input->GetGPUMemory(), (float*)params->GetGPUMemory(), numPics, inputWidth, outputWidth, numConvolutions, x1, x2, y1, y2, (input->GetSize() / inputWidth));
}

void DNNLayerConvolution::Backward(CPUGPUMemory* input, CPUGPUMemory* deltaOutput)
{
	int threadsPerBlock = 256;
	int numBlocks = ((input->GetSize() / inputWidth) + threadsPerBlock - 1) / threadsPerBlock;
	convolution_backward<<<numBlocks, threadsPerBlock>>>(deltaInput == NULL ? NULL : (float*)deltaInput->GetGPUMemory(), (float*)dparams->GetGPUMemory(), (float*)deltaOutput->GetGPUMemory(),
		(float*)output->GetGPUMemory(), (float*)input->GetGPUMemory(), (float*)params->GetGPUMemory(), numPics, inputWidth, outputWidth, numConvolutions, x1, x2, y1, y2, (input->GetSize() / inputWidth));
}
